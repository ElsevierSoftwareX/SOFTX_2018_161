#include "hip/hip_runtime.h"
 /*
This work is part of the Core Imaging Library developed by
Visual Analytics and Imaging System Group of the Science Technology
Facilities Council, STFC

Copyright 2019 Daniil Kazantsev
Copyright 2019 Srikanth Nagella, Edoardo Pasca

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/ 

#include "TGV_GPU_core.h"
#include "shared.h"

/* CUDA implementation of Primal-Dual denoising method for 
 * Total Generilized Variation (TGV)-L2 model [1] (2D/3D case)
 *
 * Input Parameters:
 * 1. Noisy image/volume (2D/3D)
 * 2. lambda - regularisation parameter
 * 3. parameter to control the first-order term (alpha1)
 * 4. parameter to control the second-order term (alpha0)
 * 5. Number of Chambolle-Pock (Primal-Dual) iterations
 * 6. Lipshitz constant (default is 12)
 *
 * Output:
 * Filtered/regularised image 
 *
 * References:
 * [1] K. Bredies "Total Generalized Variation"
 */
    
#define BLKXSIZE 8
#define BLKYSIZE 8
#define BLKZSIZE 8    
    
#define BLKXSIZE2D 8
#define BLKYSIZE2D 8
#define EPS 1.0e-7
#define idivup(a, b) ( ((a)%(b) != 0) ? (a)/(b)+1 : (a)/(b) )


/********************************************************************/
/***************************2D Functions*****************************/
/********************************************************************/
__global__ void DualP_2D_kernel(float *U, float *V1, float *V2, float *P1, float *P2, int dimX, int dimY, float sigma)
{    
	int num_total = dimX*dimY;
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        
        int index = i + dimX*j;       
        
        if (index < num_total) {  
        /* symmetric boundary conditions (Neuman) */            
        if ((i >= 0) && (i < dimX-1))  P1[index] += sigma*((U[(i+1) + dimX*j] - U[index])  - V1[index]); 
        else P1[index] += sigma*(-V1[index]); 
        if ((j >= 0) && (j < dimY-1))  P2[index] += sigma*((U[i + dimX*(j+1)] - U[index])  - V2[index]);
        else P2[index] += sigma*(-V2[index]);                    
	}
	return;
} 

__global__ void ProjP_2D_kernel(float *P1, float *P2, int dimX, int dimY, float alpha1)
{
   	float grad_magn;
	int num_total = dimX*dimY;
	
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        
        int index = i + dimX*j;
                
        if (index < num_total) {            
            grad_magn = sqrtf(pow(P1[index],2) + pow(P2[index],2));
            grad_magn = grad_magn/alpha1;
            if (grad_magn > 1.0f) {
                P1[index] /= grad_magn;
                P2[index] /= grad_magn;
            }
	}
	return;
} 

__global__ void DualQ_2D_kernel(float *V1, float *V2, float *Q1, float *Q2, float *Q3, int dimX, int dimY, float sigma)
{
        float q1, q2, q11, q22;
	int num_total = dimX*dimY;
	
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        
        int index = i + dimX*j;      
        
        if (index < num_total) {
         q1 = 0.0f; q2  = 0.0f; q11  = 0.0f; q22  = 0.0f;
         
	        if ((i >= 0) && (i < dimX-1))  {            
        	    /* boundary conditions (Neuman) */
        	    q1 = V1[(i+1) + dimX*j] - V1[index];
        	    q11 = V2[(i+1) + dimX*j] - V2[index];
	        }
        	if ((j >= 0) && (j < dimY-1)) {
        	    q2 = V2[i + dimX*(j+1)] - V2[index];
        	    q22 = V1[i + dimX*(j+1)] - V1[index];
        	}
        	
            Q1[index] += sigma*(q1);
            Q2[index] += sigma*(q2);
            Q3[index] += sigma*(0.5f*(q11 + q22));
	 }            
	return;
} 

__global__ void ProjQ_2D_kernel(float *Q1, float *Q2, float *Q3, int dimX, int dimY, float alpha0)
{
	float grad_magn;
        int num_total = dimX*dimY;

	const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        
        int index = i + dimX*j;        
       
        if (index < num_total) {
            grad_magn = sqrt(pow(Q1[index],2) + pow(Q2[index],2) + 2*pow(Q3[index],2));
            grad_magn = grad_magn/alpha0;
            if (grad_magn > 1.0f) {
                Q1[index] /= grad_magn;
                Q2[index] /= grad_magn;
                Q3[index] /= grad_magn;
        	    }
	}
	return;
} 

__global__ void DivProjP_2D_kernel(float *U, float *U0, float *P1, float *P2, int dimX, int dimY, float lambda, float tau)
{
	float P_v1, P_v2, div;
	int num_total = dimX*dimY;
	
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;       
        
        int index = i + dimX*j;

        if (index < num_total) { 
	P_v1 = 0.0f; P_v2 = 0.0f;
        
        if (i == 0) P_v1 = P1[index];
        if (i == dimX-1) P_v1 = -P1[(i-1) + dimX*j];
        if ((i > 0) && (i < dimX-1)) P_v1 = P1[index] - P1[(i-1) + dimX*j];

        if (j == 0) P_v2 = P2[index];
        if (j == dimY-1) P_v2 = -P2[i + dimX*(j-1)];
      	if ((j > 0) && (j < dimY-1))  P_v2 = P2[index] - P2[i + dimX*(j-1)];
         
        div = P_v1 + P_v2;
        U[index] = (lambda*(U[index] + tau*div) + tau*U0[index])/(lambda + tau);
	}
	return;
} 

__global__ void UpdV_2D_kernel(float *V1, float *V2, float *P1, float *P2, float *Q1, float *Q2, float *Q3, int dimX, int dimY, float tau)
{
	float q1, q3_x, q2, q3_y, div1, div2;
	int num_total = dimX*dimY;
	int i1, j1;
	
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        
        int index = i + dimX*j;      
      
	if (index < num_total) {   
	
	    i1 = (i-1) + dimX*j;
            j1 = (i) + dimX*(j-1);

            /* boundary conditions (Neuman) */        
            if ((i > 0) && (i < dimX-1)) {
            q1 = Q1[index] - Q1[i1];
            q3_x = Q3[index] - Q3[i1];  }            
            else if (i == 0) {
            q1 = Q1[index];
            q3_x = Q3[index]; } 
            else if (i == dimX-1) {
            q1 = -Q1[i1];
            q3_x = -Q3[i1];  }
            else {
            q1 = 0.0f;
            q3_x = 0.0f;
            }    
            
            if ((j > 0) && (j < dimY-1)) {
            q2 = Q2[index] - Q2[j1];
            q3_y = Q3[index] - Q3[j1]; } 
            else if (j == dimY-1) {
            q2 = -Q2[j1];
            q3_y = -Q3[j1]; }
            else if (j == 0) {
            q2 = Q2[index];
            q3_y = Q3[index]; }
            else {
            q2 = 0.0f;
            q3_y = 0.0f;
            }       
            
            div1 = q1 + q3_y;
            div2 = q3_x + q2;
            V1[index] += tau*(P1[index] + div1);
            V2[index] += tau*(P2[index] + div2);
	}
	return;
} 

__global__ void copyIm_TGV_kernel(float *U, float *U_old, int N, int M, int num_total)
{
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
    
    int index = xIndex + N*yIndex;
    
    if (index < num_total)   {
        U_old[index] = U[index];
    }
}

__global__ void copyIm_TGV_kernel_ar2(float *V1, float *V2, float *V1_old, float *V2_old, int N, int M, int num_total)
{
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
    
    int index = xIndex + N*yIndex;
    
    if (index < num_total)   {
        V1_old[index] = V1[index];
        V2_old[index] = V2[index];
    }
}

__global__ void newU_kernel(float *U, float *U_old, int N, int M, int num_total)
{
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
    
    int index = xIndex + N*yIndex;
    
    if (index < num_total)	{
        U[index] = 2.0f*U[index] - U_old[index];
    }
}


__global__ void newU_kernel_ar2(float *V1, float *V2, float *V1_old, float *V2_old, int N, int M, int num_total)
{
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
    
    int index = xIndex + N*yIndex;
    
    if (index < num_total)	{
        V1[index] = 2.0f*V1[index] - V1_old[index];
        V2[index] = 2.0f*V2[index] - V2_old[index];  
    }
}
/********************************************************************/
/***************************3D Functions*****************************/
/********************************************************************/
__global__ void DualP_3D_kernel(float *U, float *V1, float *V2, float *V3, float *P1, float *P2, float *P3, int dimX, int dimY, int dimZ, float sigma)
{    
	int index;
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int k = blockDim.z * blockIdx.z + threadIdx.z;

   	int num_total = dimX*dimY*dimZ;
        
        index = (dimX*dimY)*k + i*dimX+j;    
        if (index < num_total) {                
            /* symmetric boundary conditions (Neuman) */            
            if ((i >= 0) && (i < dimX-1)) P1[index] += sigma*((U[(dimX*dimY)*k + (i+1)*dimX+j] - U[index])  - V1[index]);  
	    else P1[index] += sigma*(-V1[index]); 
	    if ((j >= 0) && (j < dimY-1)) P2[index] += sigma*((U[(dimX*dimY)*k + i*dimX+(j+1)] - U[index])  - V2[index]);        
	    else P2[index] += sigma*(-V2[index]);                
      	    if ((k >= 0) && (k < dimZ-1)) P3[index] += sigma*((U[(dimX*dimY)*(k+1) + i*dimX+(j)] - U[index])  - V3[index]);        
	    else P3[index] += sigma*(-V3[index]);                	    
	 }	
	return;
} 

__global__ void ProjP_3D_kernel(float *P1, float *P2, float *P3, int dimX, int dimY, int dimZ, float alpha1)
{
   	float grad_magn;
   	int index;
   	int num_total = dimX*dimY*dimZ;
   	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        int k = blockDim.z * blockIdx.z + threadIdx.z;
        
        index = (dimX*dimY)*k + i*dimX+j;    
        if (index < num_total) {            
            grad_magn = (sqrtf(pow(P1[index],2) + pow(P2[index],2) + pow(P3[index],2)))/alpha1;
            if (grad_magn > 1.0f) {
                P1[index] /= grad_magn;
                P2[index] /= grad_magn;
                P3[index] /= grad_magn;
            }
	}
	return;
}

__global__ void DualQ_3D_kernel(float *V1, float *V2, float *V3, float *Q1, float *Q2, float *Q3, float *Q4, float *Q5, float *Q6, int dimX, int dimY, int dimZ, float sigma)
{
	int index; 
        float q1, q2, q3, q11, q22, q33, q44, q55, q66;

   	int num_total = dimX*dimY*dimZ;
   	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        int k = blockDim.z * blockIdx.z + threadIdx.z;
        
        index = (dimX*dimY)*k + i*dimX+j; 
        int i1 = (dimX*dimY)*k + (i+1)*dimX+j;
        int j1 = (dimX*dimY)*k + (i)*dimX+(j+1);
        int k1 = (dimX*dimY)*(k+1) + (i)*dimX+(j);
                
        if (index < num_total) {
 	q1 = 0.0f; q11 = 0.0f; q33 = 0.0f; q2 = 0.0f; q22 = 0.0f; q55 = 0.0f; q3 = 0.0f; q44 = 0.0f; q66 = 0.0f;
         
	        /* boundary conditions (Neuman) */
	        if ((i >= 0) && (i < dimX-1))  {                    	
                q1 = V1[i1] - V1[index];              
                q11 = V2[i1] - V2[index];
                q33 = V3[i1] - V3[index];  }
        	if ((j >= 0) && (j < dimY-1)) {
                q2 = V2[j1] - V2[index];                
                q22 = V1[j1] - V1[index];
                q55 = V3[j1] - V3[index];  }
        	if ((k >= 0) && (k < dimZ-1)) {
                q3 = V3[k1] - V3[index];
                q44 = V1[k1] - V1[index];
                q66 = V2[k1] - V2[index]; }
        	
            Q1[index] += sigma*(q1); /*Q11*/
            Q2[index] += sigma*(q2); /*Q22*/            
            Q3[index] += sigma*(q3); /*Q33*/
            Q4[index] += sigma*(0.5f*(q11 + q22)); /* Q21 / Q12 */
            Q5[index] += sigma*(0.5f*(q33 + q44)); /* Q31 / Q13 */
            Q6[index] += sigma*(0.5f*(q55 + q66)); /* Q32 / Q23 */
	 }
	return;
}

__global__ void ProjQ_3D_kernel(float *Q1, float *Q2, float *Q3, float *Q4, float *Q5, float *Q6, int dimX, int dimY, int dimZ, float alpha0)
{
	float grad_magn;
	int index;
   	int num_total = dimX*dimY*dimZ;
   	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        int k = blockDim.z * blockIdx.z + threadIdx.z;        

        index = (dimX*dimY)*k + i*dimX+j; 
        
        if (index < num_total) {	
	grad_magn = sqrtf(pow(Q1[index],2) + pow(Q2[index],2) + pow(Q3[index],2) + 2.0f*pow(Q4[index],2) + 2.0f*pow(Q5[index],2) + 2.0f*pow(Q6[index],2));
            grad_magn = grad_magn/alpha0;
            if (grad_magn > 1.0f) {
                Q1[index] /= grad_magn;
                Q2[index] /= grad_magn;
                Q3[index] /= grad_magn;
                Q4[index] /= grad_magn;
                Q5[index] /= grad_magn;
                Q6[index] /= grad_magn;
            }
	}
	return;
} 
__global__ void DivProjP_3D_kernel(float *U, float *U0, float *P1, float *P2, float *P3, int dimX, int dimY, int dimZ, float lambda, float tau)
{
	float P_v1, P_v2, P_v3, div;
	int index;
   	int num_total = dimX*dimY*dimZ;
   	
   	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        int k = blockDim.z * blockIdx.z + threadIdx.z;       
     
        index = (dimX*dimY)*k + i*dimX+j; 
        int i1 = (dimX*dimY)*k + (i-1)*dimX+j;
        int j1 = (dimX*dimY)*k + (i)*dimX+(j-1);
        int k1 = (dimX*dimY)*(k-1) + (i)*dimX+(j);        
        
        if (index < num_total) { 
	P_v1 = 0.0f; P_v2 = 0.0f; P_v3 = 0.0f;
        
        if (i == 0) P_v1 = P1[index];
        if (i == dimX-1) P_v1 = -P1[i1];
        if ((i > 0) && (i < dimX-1)) P_v1 = P1[index] - P1[i1];

        if (j == 0) P_v2 = P2[index];
        if (j == dimY-1) P_v2 = -P2[j1];
      	if ((j > 0) && (j < dimY-1))  P_v2 = P2[index] - P2[j1];
        
        if (k == 0) P_v3 = P3[index];
        if (k == dimZ-1) P_v3 = -P3[k1];
      	if ((k > 0) && (k < dimZ-1))  P_v3 = P3[index] - P3[k1];
     
                      
        div = P_v1 + P_v2 + P_v3;
        U[index] = (lambda*(U[index] + tau*div) + tau*U0[index])/(lambda + tau);             
	}
	return;
}
__global__ void UpdV_3D_kernel(float *V1, float *V2, float *V3, float *P1, float *P2, float *P3, float *Q1, float *Q2, float *Q3, float *Q4, float *Q5, float *Q6, int dimX, int dimY, int dimZ, float tau)
{
	float q1, q4x, q5x, q2, q4y, q6y, q6z, q5z, q3, div1, div2, div3;
	int index;
	int num_total = dimX*dimY*dimZ;
	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        int k = blockDim.z * blockIdx.z + threadIdx.z;
        
        index = (dimX*dimY)*k + i*dimX+j; 
        int i1 = (dimX*dimY)*k + (i-1)*dimX+j;
        int j1 = (dimX*dimY)*k + (i)*dimX+(j-1);
        int k1 = (dimX*dimY)*(k-1) + (i)*dimX+(j);    
        
        /* Q1 - Q11, Q2 - Q22, Q3 -  Q33, Q4 - Q21/Q12, Q5 - Q31/Q13, Q6 - Q32/Q23*/       
        if (index < num_total) {  

  	 /* boundary conditions (Neuman) */        
            if ((i > 0) && (i < dimX-1)) {
                q1 = Q1[index] - Q1[i1];
                q4x = Q4[index] - Q4[i1];                
                q5x = Q5[index] - Q5[i1]; }            
            else if (i == 0) {
                q1 = Q1[index];
                q4x = Q4[index];                
                q5x = Q5[index]; } 
            else if (i == dimX-1) {
                q1 = -Q1[i1];
                q4x = -Q4[i1];                
                q5x = -Q5[i1]; }
            else {
                q1 = 0.0f;
                q4x = 0.0f;
                q5x = 0.0f;  }    
            
            if ((j > 0) && (j < dimY-1)) {
                q2 = Q2[index] - Q2[j1];
                q4y = Q4[index] - Q4[j1];
                q6y = Q6[index] - Q6[j1]; } 
            else if (j == dimY-1) {
                q2 = -Q2[j1];
                q4y = -Q4[j1];
                q6y = -Q6[j1]; }
            else if (j == 0) {
                q2 = Q2[index];
                q4y = Q4[index];
                q6y = Q6[index]; }
            else {
                q2 =  0.0f;
                q4y = 0.0f;
                q6y = 0.0f;
               }       

            if ((k > 0) && (k < dimZ-1)) {
                q6z = Q6[index] - Q6[k1];
                q5z = Q5[index] - Q5[k1];
                q3 = Q3[index] - Q3[k1]; } 
            else if (k == dimZ-1) {
                q6z = -Q6[k1];
                q5z = -Q5[k1];
                q3 = -Q3[k1]; }
            else if (k == 0) {
                q6z = Q6[index];
                q5z = Q5[index];
                q3 = Q3[index]; }
            else {
                q6z = 0.0f;
                q5z = 0.0f;
                q3 = 0.0f; }

       div1 = q1 + q4y + q5z;
       div2 = q4x + q2 + q6z;            
       div3 = q5x + q6y + q3;
            
        V1[index] += tau*(P1[index] + div1);
        V2[index] += tau*(P2[index] + div2);
        V3[index] += tau*(P3[index] + div3);
	}
	return;
} 

__global__ void copyIm_TGV_kernel3D(float *U, float *U_old, int dimX, int dimY, int dimZ, int num_total)
{
    int index;
	
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;    
    
    index = (dimX*dimY)*k + j*dimX+i;
    
    if (index < num_total) {	
      	U_old[index] = U[index];	
    }
}

__global__ void copyIm_TGV_kernel3D_ar3(float *V1, float *V2, float *V3, float *V1_old, float *V2_old, float *V3_old, int dimX, int dimY, int dimZ, int num_total)
{
    int index;
	
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;    
    
    index = (dimX*dimY)*k + j*dimX+i;
    
    if (index < num_total) {	
      	V1_old[index] = V1[index];
	V2_old[index] = V2[index];
	V3_old[index] = V3[index];	
    }
}

__global__ void newU_kernel3D(float *U, float *U_old, int dimX, int dimY, int dimZ, int num_total)
{
     int index;
	
     int i = blockDim.x * blockIdx.x + threadIdx.x;
     int j = blockDim.y * blockIdx.y + threadIdx.y;
     int k = blockDim.z * blockIdx.z + threadIdx.z;    
         
     index = (dimX*dimY)*k + j*dimX+i;
    
    if (index < num_total) {
	   U[index] = 2.0f*U[index] - U_old[index];
    }
}  

__global__ void newU_kernel3D_ar3(float *V1, float *V2, float *V3, float *V1_old, float *V2_old, float *V3_old, int dimX, int dimY, int dimZ, int num_total)
{
     int index;
	
     int i = blockDim.x * blockIdx.x + threadIdx.x;
     int j = blockDim.y * blockIdx.y + threadIdx.y;
     int k = blockDim.z * blockIdx.z + threadIdx.z;    
         
     index = (dimX*dimY)*k + j*dimX+i;
    
    if (index < num_total) {
	   V1[index] = 2.0f*V1[index] - V1_old[index];
	   V2[index] = 2.0f*V2[index] - V2_old[index];
	   V3[index] = 2.0f*V3[index] - V3_old[index];
    }
}  

/*%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%*/
/************************ MAIN HOST FUNCTION ***********************/
/*%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%*/
extern "C" int TGV_GPU_main(float *U0, float *U, float lambda, float alpha1, float alpha0, int iterationsNumb, float L2, int dimX, int dimY, int dimZ)
{
	int dimTotal, dev = 0;
	CHECK(hipSetDevice(dev));
	
	dimTotal = dimX*dimY*dimZ;
       
        float *U_old, *d_U0, *d_U, *P1, *P2, *Q1, *Q2, *Q3, *V1, *V1_old, *V2, *V2_old, tau, sigma;
        tau = pow(L2,-0.5);
        sigma = pow(L2,-0.5);
                                      
        CHECK(hipMalloc((void**)&d_U0,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&d_U,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&U_old,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&P1,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&P2,dimTotal*sizeof(float)));
        
        CHECK(hipMalloc((void**)&Q1,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&Q2,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&Q3,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&V1,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&V2,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&V1_old,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&V2_old,dimTotal*sizeof(float)));
        
        CHECK(hipMemcpy(d_U0,U0,dimTotal*sizeof(float),hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_U,U0,dimTotal*sizeof(float),hipMemcpyHostToDevice));   
        hipMemset(P1, 0, dimTotal*sizeof(float));
        hipMemset(P2, 0, dimTotal*sizeof(float));
        hipMemset(Q1, 0, dimTotal*sizeof(float));
        hipMemset(Q2, 0, dimTotal*sizeof(float));
        hipMemset(Q3, 0, dimTotal*sizeof(float));
        hipMemset(V1, 0, dimTotal*sizeof(float));
        hipMemset(V2, 0, dimTotal*sizeof(float));           
        
        if (dimZ == 1) {
	/*2D case */
        dim3 dimBlock(BLKXSIZE2D,BLKYSIZE2D);
        dim3 dimGrid(idivup(dimX,BLKXSIZE2D), idivup(dimY,BLKYSIZE2D));
             
        for(int n=0; n < iterationsNumb; n++) {
			
	    /* Calculate Dual Variable P */
            DualP_2D_kernel<<<dimGrid,dimBlock>>>(d_U, V1, V2, P1, P2, dimX, dimY, sigma);
      	    CHECK(hipDeviceSynchronize());
            /*Projection onto convex set for P*/
            ProjP_2D_kernel<<<dimGrid,dimBlock>>>(P1, P2, dimX, dimY, alpha1);
            CHECK(hipDeviceSynchronize());
            /* Calculate Dual Variable Q */
            DualQ_2D_kernel<<<dimGrid,dimBlock>>>(V1, V2, Q1, Q2, Q3, dimX, dimY, sigma);
            CHECK(hipDeviceSynchronize());
            /*Projection onto convex set for Q*/
            ProjQ_2D_kernel<<<dimGrid,dimBlock>>>(Q1, Q2, Q3, dimX, dimY, alpha0);
            CHECK(hipDeviceSynchronize());
            /*saving U into U_old*/
            copyIm_TGV_kernel<<<dimGrid,dimBlock>>>(d_U, U_old, dimX, dimY, dimTotal);
            CHECK(hipDeviceSynchronize());
            /*adjoint operation  -> divergence and projection of P*/
            DivProjP_2D_kernel<<<dimGrid,dimBlock>>>(d_U, d_U0, P1, P2, dimX, dimY, lambda, tau);
            CHECK(hipDeviceSynchronize());
            /*get updated solution U*/
            newU_kernel<<<dimGrid,dimBlock>>>(d_U, U_old, dimX, dimY, dimTotal);
            CHECK(hipDeviceSynchronize());
            /*saving V into V_old*/
            copyIm_TGV_kernel_ar2<<<dimGrid,dimBlock>>>(V1, V2, V1_old, V2_old, dimX, dimY, dimTotal);
            CHECK(hipDeviceSynchronize());
            /* upd V*/
            UpdV_2D_kernel<<<dimGrid,dimBlock>>>(V1, V2, P1, P2, Q1, Q2, Q3, dimX, dimY, tau);
            CHECK(hipDeviceSynchronize());
            /*get new V*/
            newU_kernel_ar2<<<dimGrid,dimBlock>>>(V1, V2, V1_old, V2_old, dimX, dimY, dimTotal);
            CHECK(hipDeviceSynchronize());            
	    }
        }
        else {
        /*3D case */
        dim3 dimBlock(BLKXSIZE,BLKYSIZE,BLKZSIZE);
        dim3 dimGrid(idivup(dimX,BLKXSIZE), idivup(dimY,BLKYSIZE),idivup(dimZ,BLKXSIZE));
        
        float *P3, *Q4, *Q5, *Q6, *V3, *V3_old;
        
	CHECK(hipMalloc((void**)&P3,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&Q4,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&Q5,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&Q6,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&V3,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&V3_old,dimTotal*sizeof(float)));
        
        hipMemset(Q4, 0.0f, dimTotal*sizeof(float));
        hipMemset(Q5, 0.0f, dimTotal*sizeof(float));
        hipMemset(Q6, 0.0f, dimTotal*sizeof(float));
        hipMemset(P3, 0.0f, dimTotal*sizeof(float));
        hipMemset(V3, 0.0f, dimTotal*sizeof(float));        
        
        for(int n=0; n < iterationsNumb; n++) {
			
	    /* Calculate Dual Variable P */
            DualP_3D_kernel<<<dimGrid,dimBlock>>>(d_U, V1, V2, V3, P1, P2, P3, dimX, dimY, dimZ, sigma);
	    CHECK(hipDeviceSynchronize());
            /*Projection onto convex set for P*/
            ProjP_3D_kernel<<<dimGrid,dimBlock>>>(P1, P2, P3, dimX, dimY, dimZ, alpha1);
            CHECK(hipDeviceSynchronize());
            /* Calculate Dual Variable Q */
            DualQ_3D_kernel<<<dimGrid,dimBlock>>>(V1, V2, V3, Q1, Q2, Q3, Q4, Q5, Q6, dimX, dimY, dimZ, sigma);
            CHECK(hipDeviceSynchronize());
             /*Projection onto convex set for Q*/
            ProjQ_3D_kernel<<<dimGrid,dimBlock>>>(Q1, Q2, Q3, Q4, Q5, Q6, dimX, dimY, dimZ, alpha0);
            CHECK(hipDeviceSynchronize());
            /*saving U into U_old*/
            copyIm_TGV_kernel3D<<<dimGrid,dimBlock>>>(d_U, U_old, dimX, dimY, dimZ, dimTotal);
            CHECK(hipDeviceSynchronize());
            /*adjoint operation  -> divergence and projection of P*/
            DivProjP_3D_kernel<<<dimGrid,dimBlock>>>(d_U, d_U0, P1, P2, P3, dimX, dimY, dimZ, lambda, tau);
            CHECK(hipDeviceSynchronize());
            /*get updated solution U*/
            newU_kernel3D<<<dimGrid,dimBlock>>>(d_U, U_old, dimX, dimY, dimZ, dimTotal);
            CHECK(hipDeviceSynchronize());
            /*saving V into V_old*/
            copyIm_TGV_kernel3D_ar3<<<dimGrid,dimBlock>>>(V1, V2, V3, V1_old, V2_old, V3_old, dimX, dimY, dimZ, dimTotal);           
            CHECK(hipDeviceSynchronize());
            /* upd V*/
            UpdV_3D_kernel<<<dimGrid,dimBlock>>>(V1, V2, V3, P1, P2, P3, Q1, Q2, Q3, Q4, Q5, Q6, dimX, dimY, dimZ, tau);
            CHECK(hipDeviceSynchronize());
            /*get new V*/
            newU_kernel3D_ar3<<<dimGrid,dimBlock>>>(V1, V2, V3, V1_old, V2_old, V3_old, dimX, dimY, dimZ, dimTotal);
            CHECK(hipDeviceSynchronize());            
	        }
	        
        CHECK(hipFree(Q4));
        CHECK(hipFree(Q5));
        CHECK(hipFree(Q6));
        CHECK(hipFree(P3));
        CHECK(hipFree(V3));
        CHECK(hipFree(V3_old));	                
        }
        
        CHECK(hipMemcpy(U,d_U,dimTotal*sizeof(float),hipMemcpyDeviceToHost));
        CHECK(hipFree(d_U0));
        CHECK(hipFree(d_U));
        CHECK(hipFree(U_old));
        CHECK(hipFree(P1));
        CHECK(hipFree(P2));
        
        CHECK(hipFree(Q1));
        CHECK(hipFree(Q2));
        CHECK(hipFree(Q3));
        CHECK(hipFree(V1));
        CHECK(hipFree(V2));
        CHECK(hipFree(V1_old));
        CHECK(hipFree(V2_old));
        return 0;
}

#include "hip/hip_runtime.h"
 /*
This work is part of the Core Imaging Library developed by
Visual Analytics and Imaging System Group of the Science Technology
Facilities Council, STFC

Copyright 2017 Daniil Kazantsev
Copyright 2017 Srikanth Nagella, Edoardo Pasca

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/ 

#include "NonlDiff_GPU_core.h"

/* CUDA implementation of linear and nonlinear diffusion with the regularisation model [1,2] (2D/3D case)
 * The minimisation is performed using explicit scheme. 
 *
 * Input Parameters:
 * 1. Noisy image/volume 
 * 2. lambda - regularization parameter
 * 3. Edge-preserving parameter (sigma), when sigma equals to zero nonlinear diffusion -> linear diffusion
 * 4. Number of iterations, for explicit scheme >= 150 is recommended 
 * 5. tau - time-marching step for explicit scheme
 * 6. Penalty type: 1 - Huber, 2 - Perona-Malik, 3 - Tukey Biweight
 *
 * Output:
 * [1] Regularized image/volume 
 *
 * This function is based on the paper by
 * [1] Perona, P. and Malik, J., 1990. Scale-space and edge detection using anisotropic diffusion. IEEE Transactions on pattern analysis and machine intelligence, 12(7), pp.629-639.
 * [2] Black, M.J., Sapiro, G., Marimont, D.H. and Heeger, D., 1998. Robust anisotropic diffusion. IEEE Transactions on image processing, 7(3), pp.421-432.
 */


#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}
    
#define BLKXSIZE 8
#define BLKYSIZE 8
#define BLKZSIZE 8
    
#define BLKXSIZE2D 16
#define BLKYSIZE2D 16
#define EPS 1.0e-5
    
#define idivup(a, b) ( ((a)%(b) != 0) ? (a)/(b)+1 : (a)/(b) )

#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__host__ __device__ int sign (float x)
{
        return (x > 0) - (x < 0);
}        
   
/********************************************************************/
/***************************2D Functions*****************************/
/********************************************************************/
__global__ void LinearDiff2D_kernel(float *Input, float *Output, float lambdaPar, float tau, int N, int M)
    {
		int i1,i2,j1,j2;
		float e,w,n,s,e1,w1,n1,s1;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        
        int index = i + N*j;
        
        if ((i >= 0) && (i < N) && (j >= 0) && (j < M)) {
            
            /* boundary conditions (Neumann reflections) */
			i1 = i+1; if (i1 == N) i1 = i-1;
			i2 = i-1; if (i2 < 0) i2 = i+1;
            j1 = j+1; if (j1 == M) j1 = j-1;
            j2 = j-1; if (j2 < 0) j2 = j+1;
            
		        e = Output[j*N+i1];
                w = Output[j*N+i2];
                n = Output[j1*N+i];
                s = Output[j2*N+i];
                
                e1 = e - Output[index];
                w1 = w - Output[index];
                n1 = n - Output[index];
                s1 = s - Output[index];
                
                Output[index] += tau*(lambdaPar*(e1 + w1 + n1 + s1) - (Output[index] - Input[index])); 
		}
	} 
    
 __global__ void NonLinearDiff2D_kernel(float *Input, float *Output, float lambdaPar, float sigmaPar, float tau, int penaltytype, int N, int M)
    {
		int i1,i2,j1,j2;
		float e,w,n,s,e1,w1,n1,s1;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        
        int index = i + N*j;
        
        if ((i >= 0) && (i < N) && (j >= 0) && (j < M)) {
            
            /* boundary conditions (Neumann reflections) */
			i1 = i+1; if (i1 == N) i1 = i-1;
			i2 = i-1; if (i2 < 0) i2 = i+1;
            j1 = j+1; if (j1 == M) j1 = j-1;
            j2 = j-1; if (j2 < 0) j2 = j+1;
            
		        e = Output[j*N+i1];
                w = Output[j*N+i2];
                n = Output[j1*N+i];
                s = Output[j2*N+i];
                
                e1 = e - Output[index];
                w1 = w - Output[index];
                n1 = n - Output[index];
                s1 = s - Output[index];
                
            if (penaltytype == 1){
            /* Huber penalty */
            if (abs(e1) > sigmaPar) e1 =  sign(e1);
            else e1 = e1/sigmaPar;
            
            if (abs(w1) > sigmaPar) w1 =  sign(w1);
            else w1 = w1/sigmaPar;
            
            if (abs(n1) > sigmaPar) n1 =  sign(n1);
            else n1 = n1/sigmaPar;
            
            if (abs(s1) > sigmaPar) s1 =  sign(s1);
            else s1 = s1/sigmaPar;
            }
            else if (penaltytype == 2) {
            /* Perona-Malik */
            e1 = (e1)/(1.0f + pow((e1/sigmaPar),2));
            w1 = (w1)/(1.0f + pow((w1/sigmaPar),2));
            n1 = (n1)/(1.0f + pow((n1/sigmaPar),2));
            s1 = (s1)/(1.0f + pow((s1/sigmaPar),2));
            }
            else if (penaltytype == 3) {
            /* Tukey Biweight */
            if (abs(e1) <= sigmaPar) e1 =  e1*pow((1.0f - pow((e1/sigmaPar),2)), 2);
            else e1 = 0.0f;
            if (abs(w1) <= sigmaPar) w1 =  w1*pow((1.0f - pow((w1/sigmaPar),2)), 2);
            else w1 = 0.0f;
            if (abs(n1) <= sigmaPar) n1 =  n1*pow((1.0f - pow((n1/sigmaPar),2)), 2);
            else n1 = 0.0f;
            if (abs(s1) <= sigmaPar) s1 =  s1*pow((1.0f - pow((s1/sigmaPar),2)), 2);
            else s1 = 0.0f;
            }
            else printf("%s \n", "No penalty function selected! Use 1,2 or 3.");
                            
            Output[index] += tau*(lambdaPar*(e1 + w1 + n1 + s1) - (Output[index] - Input[index])); 
		}
	} 
/********************************************************************/
/***************************3D Functions*****************************/
/********************************************************************/

/////////////////////////////////////////////////
// HOST FUNCTION
extern "C" void NonlDiff_GPU_main(float *Input, float *Output, float lambdaPar, float sigmaPar, int iterationsNumb, float tau, int penaltytype, int N, int M, int Z)
{
	    // set up device
		int dev = 0;
		CHECK(hipSetDevice(dev));
        float *d_input, *d_output;
        float sigmaPar2;
        sigmaPar2 = sigmaPar/sqrt(2.0f);
        
	if (Z == 1) {
	     /*2D case */ 
        CHECK(hipMalloc((void**)&d_input,N*M*Z*sizeof(float)));
        CHECK(hipMalloc((void**)&d_output,N*M*Z*sizeof(float)));
                
        CHECK(hipMemcpy(d_input,Input,N*M*Z*sizeof(float),hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_output,Input,N*M*Z*sizeof(float),hipMemcpyHostToDevice));      
        
        dim3 dimBlock(BLKXSIZE2D,BLKYSIZE2D);
        dim3 dimGrid(idivup(N,BLKXSIZE2D), idivup(M,BLKYSIZE2D));
             
        for(int n=0; n < iterationsNumb; n++) {
				if (sigmaPar == 0.0f) {
				/* linear diffusion (heat equation) */
				LinearDiff2D_kernel<<<dimGrid,dimBlock>>>(d_input, d_output, lambdaPar, tau, N, M);
				CHECK(hipDeviceSynchronize());
				}
				else {
				/* nonlinear diffusion */
				NonLinearDiff2D_kernel<<<dimGrid,dimBlock>>>(d_input, d_output, lambdaPar, sigmaPar2, tau, penaltytype, N, M);
				CHECK(hipDeviceSynchronize());
				}
        }
	}
	else {
		/*3D case*/
		}
        
        CHECK(hipMemcpy(Output,d_output,N*M*Z*sizeof(float),hipMemcpyDeviceToHost));
        CHECK(hipFree(d_input));
        CHECK(hipFree(d_output));
        hipDeviceReset(); 
}
